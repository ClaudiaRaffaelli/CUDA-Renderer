#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaRenderer.h"
#include "image.h"
#include "sceneLoader.h"

//Defining some constants

struct GlobalConstants {

    SceneName sceneName;

    int numCircles;
    float* position;
    float* color;
    float* radius;

    int imageWidth;
    int imageHeight;
    float* imageData;
};

// Global variable that is in scope, but read-only, for all CUDA
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (constant memory
// is a fast place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;

//Constants for hardware: GPU GeForge GT 630M (compute capability 2.1)
#define THREADS_PER_BLOCK_X 32
#define THREADS_PER_BLOCK_Y 32
//SCAN_BLOCK_DIM=1024 max number of threads per block for this hardware.
//Is used the name SCAN_BLOCK_DIM because of the file exclusiveScan.cu_inl
#define SCAN_BLOCK_DIM (THREADS_PER_BLOCK_X * THREADS_PER_BLOCK_Y)


//Including others useful utilities
#include "util.h"
#include "circleBoxTest.cu_inl"
#include "exclusiveScan.cu_inl"


////////////////////////////////////////////////////////////////////////////////////////
// Putting all the CUDA kernels here
///////////////////////////////////////////////////////////////////////////////////////


// kernelClearImage --  (CUDA device code)
//
// Clear the image, setting all pixels to the specified color rgba
__global__ void kernelClearImage(float r, float g, float b, float a) {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r, g, b, a);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four separate fp32 stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}


// shadePixel -- (CUDA device code)
//
// given a pixel and a circle, determines the contribution to the
// pixel from the circle.  Update of the image is done in this
// function.  Called by kernelRenderCircles()
// inline function: increases compile time but saves a lot of time in runtime
__device__ __inline__ void
shadePixel(int circleIndex, float2 pixelCenter, float3 p, float4* imagePtr) {

    float diffX = p.x - pixelCenter.x;
    float diffY = p.y - pixelCenter.y;
    float pixelDist = diffX * diffX + diffY * diffY;

    float rad = cuConstRendererParams.radius[circleIndex];
    float maxDist = rad * rad;

    // circle does not contribute to the image
    if (pixelDist > maxDist)
        return;

    float3 rgb;
    float alpha;

    // there is a non-zero contribution.  Now compute the shading value

    // simple: each circle has an assigned color
    int index3 = 3 * circleIndex;
    rgb = *(float3*)&(cuConstRendererParams.color[index3]);
    alpha = .5f;


    float oneMinusAlpha = 1.f - alpha;

    // BEGIN SHOULD-BE-ATOMIC REGION
    // global memory read

    float4 existingColor = *imagePtr;
    float4 newColor;
    newColor.x = alpha * rgb.x + oneMinusAlpha * existingColor.x;
    newColor.y = alpha * rgb.y + oneMinusAlpha * existingColor.y;
    newColor.z = alpha * rgb.z + oneMinusAlpha * existingColor.z;
    newColor.w = alpha + existingColor.w;

    // global memory write
    *imagePtr = newColor;

    // END SHOULD-BE-ATOMIC REGION
}

//Function used by each block to compute the number of circles present in the current
//block. Every block gets a small portion of the original image of
//dimension THREADS_PER_BLOCK_X * THREADS_PER_BLOCK_Y = SCAN_BLOCK_DIM pixels
__device__ __inline__ uint
countCircles(short * blockCoord, uint * circleCountPerThreadList, uint * circleIndexesInBlockList, uint * circleCountPerBlockList){

	// Computing the thread index for that block (as required from exclusiveScan.cu_inl)
	int linearThreadIndex = threadIdx.y * blockDim.x + threadIdx.x;

	short imageWidth = cuConstRendererParams.imageWidth;
	short imageHeight = cuConstRendererParams.imageHeight;

	//inverted width and height
	float invWidth = 1.f / imageWidth;
	float invHeight = 1.f / imageHeight;


	//Remembering that blockCoord[0] is the block left coordinate,
	//[1] the right coordinate, [2] the top, and [3] the bottom

	// array of block coordinates normalized
	float leftIndex = blockCoord[0] * invWidth;
	float rightIndex = blockCoord[1] * invWidth;
	float topIndex = blockCoord[2] * invHeight;
	float bottomIndex = blockCoord[3] * invHeight;

	//Each thread inside the block handles one portion of the array containing all the
	//circles within the image. Each of this threads check if their circles are
	//present or not in the block. When all the threads have finished (after the
	//syncthreads()) we have all the circles within the block.

	//Each thread gets an equal amount of circle index to handle, depending on how many
	//circles there are to draw in the original image.

	int circlesPerThread = (cuConstRendererParams.numCircles + SCAN_BLOCK_DIM - 1) / SCAN_BLOCK_DIM;
	int circleIndexStart = linearThreadIndex * circlesPerThread;
	int circleIndexEnd=0;
	if(linearThreadIndex == SCAN_BLOCK_DIM)
		circleIndexEnd = cuConstRendererParams.numCircles;
	else{
		circleIndexEnd = circleIndexStart + circlesPerThread;
	}

	int circleCountPerThread = 0;

	//CIRCLES_PER_THREAD must contain the size of the array circleArrayPerThread.
	//We don't know yet how many circles there will be in a certain block.
	//The idea is to take a number big enough
	const uint CIRCLES_PER_THREAD=32;
	uint circleArrayPerThread[CIRCLES_PER_THREAD];

	for(int i = circleIndexStart; i< circleIndexEnd; i++){
		if(i<cuConstRendererParams.numCircles){
			float3 position = *(float3*)(&cuConstRendererParams.position[i*3]);
			float radius = cuConstRendererParams.radius[i];

			//call to func from file circleBoxTest.cu_inl to determine if the bounding box of the
			//circle belongs for at least one pixel to the current block
			if(circleInBoxConservative(position.x, position.y, radius, leftIndex, rightIndex, bottomIndex, topIndex) == 1){
				circleArrayPerThread[circleCountPerThread] = i;
				circleCountPerThread++;
			}
		}
	}

	// circleCount contains for each thread the number of circles within that block, found handling
	// a portion of the original array of all circles within the image.

	circleCountPerThreadList[ linearThreadIndex ] = circleCountPerThread;
	__syncthreads();

	//Using the utility given by the file exlusiveScan.cu_inl is possible to compute the sum of each
	//element of the array circleCount. By doing so we have now the total amount of circles present
	//within the current block

	sharedMemExclusiveScan(linearThreadIndex, circleCountPerThreadList, circleCountPerBlockList, circleIndexesInBlockList, SCAN_BLOCK_DIM);
	__syncthreads();

	//updating the totalCircles (sharedMemExlusiveScan is exclusive, so we have to add the last element)
	uint totalCircles = circleCountPerBlockList[SCAN_BLOCK_DIM-1] + circleCountPerThreadList[SCAN_BLOCK_DIM-1];

	//Before returning we have to copy back the list of index for each block (keeping the ordering unchanged)
	uint tmpIndex = circleCountPerBlockList[ linearThreadIndex ];

	for(int i=0; i<circleCountPerThread; i++){
		circleIndexesInBlockList[tmpIndex] = circleArrayPerThread[i];
		tmpIndex++;
	}
	__syncthreads();

	return totalCircles;
}


// kernelRenderCircles -- (CUDA device code)
//
// The image is divided in smaller fractions treated individually
// by computing the number of circles in that block.
// After that, knowing the number and index of circles per block
// is possible to "shade" all the pixels of the block in a parallel
// way by assigning each pixel to a different thread. Each thread
// runs through the list of circles per block (in which the ordering
// is correct), check if that pixel belongs to the circle. If so
// it applies the shading to that pixel, otherwise pass over.
__global__ void kernelRenderCircles() {

	//Part 1:
	//Consists in dividing the image in small parts (one for each
	//block) that are handled individually. First of all are computed how many
	//and which circles are present per block.
	//Each thread in a block works on a different portion of the array of circles,
	//so that by the end of the computation the entire list of circles has been analyzed.
	//(Only if there are many circles in the image, otherwise this step is avoided)

	//For the task of finding what circles are present in a block, are needed three
	//arrays in shared memory (shared between the threads of the same block), used
	//by the file exclusiveScan.cu_inl for:

	//storing for each thread how many circles has found, each of them working on
	//a small portion of the original array with all the circles in the image.
	__shared__ uint circleCountPerThreadList[SCAN_BLOCK_DIM];
	//storing the number of circles found per block
	__shared__ uint circleCountPerBlockList[SCAN_BLOCK_DIM];
	//keeping the circle indexes found in the block
	__shared__ uint circleIndexesInBlockList[SCAN_BLOCK_DIM * 2];

	short imageWidth = cuConstRendererParams.imageWidth;
	short imageHeight = cuConstRendererParams.imageHeight;

	//inverted width and height
	float invWidth = 1.f / imageWidth;
	float invHeight = 1.f / imageHeight;

	//Setting the block coordinates

	short blockLeftCoord = blockIdx.x * THREADS_PER_BLOCK_X;
	short blockRightCoord = blockLeftCoord + THREADS_PER_BLOCK_X - 1;
	short blockTopCoord = blockIdx.y * THREADS_PER_BLOCK_Y;
	short blockBottomCoord = blockTopCoord + THREADS_PER_BLOCK_Y - 1;


	short blockCoord[]{blockLeftCoord,blockRightCoord,blockTopCoord,blockBottomCoord };

	//Compute the list of circles regarding the current block.
	//The functions returns the total number of circles within the current
	//block. Also it sets circleCount, circleIndexesInBlock and indexList (?)
	uint circlesNumberTotal=countCircles(blockCoord, circleCountPerThreadList, circleIndexesInBlockList, circleCountPerBlockList);

	//Part 2:
	//circlesNumberTotal contains the number of circles within the block.
	//In this part is performed the rendering. Each thread gets the coordinate
	//of a pixel. Then for each pixel is checked if that pixel belongs or not
	//to the first circle in the restricted list. If so, the color of the pixel is
	//updated, otherwise not. In both cases the next move is to check the second
	//circle and so on.

	// Computing the pixels coordinates
	uint pixelXCoord = blockLeftCoord + threadIdx.x;
	uint pixelYCoord = blockTopCoord + threadIdx.y;

	// Computed imgPtr and the pixel center
	float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[4 * (pixelYCoord * imageWidth + pixelXCoord)]);
	float2 pixelCenterNorm = make_float2(invWidth * (static_cast<float>(pixelXCoord) + 0.5f),
	    invHeight * (static_cast<float>(pixelYCoord) + 0.5f));

	// Given a pixel is checked for all the circles counted within the block if that
	//pixel belongs to the circle or not, by calling shadePixel. shadePixel is also responsible
	//for coloring properly the pixel.
	//The right coloring order is respected because of the way the circleIndexesInBlock is created. The list
	//is restricted to the circles present within the block but with the same ordering.
	for(uint i=0; i<circlesNumberTotal;i++){
		uint circleIndex=circleIndexesInBlockList[i];
		float3 circlePosition=*(float3*)(&cuConstRendererParams.position[circleIndex*3]);
		shadePixel(circleIndex, pixelCenterNorm, circlePosition, imgPtr);
	}

}

////////////////////////////////////////////////////////////////////////////////////////


CudaRenderer::CudaRenderer() {
    image = NULL;

    numCircles = 0;
    position = NULL;
    color = NULL;
    radius = NULL;

    cudaDevicePosition = NULL;
    cudaDeviceColor = NULL;
    cudaDeviceRadius = NULL;
    cudaDeviceImageData = NULL;
}

CudaRenderer::~CudaRenderer() {

    if (image) {
        delete image;
    }

    if (position) {
        delete [] position;
        delete [] color;
        delete [] radius;
    }

    if (cudaDevicePosition) {
        hipFree(cudaDevicePosition);
        hipFree(cudaDeviceColor);
        hipFree(cudaDeviceRadius);
        hipFree(cudaDeviceImageData);
    }
}

const Image*
CudaRenderer::getImage() {

    // need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller

    printf("Copying image data from device\n");

    hipMemcpy(image->data,
               cudaDeviceImageData,
               sizeof(float) * 4 * image->width * image->height,
               hipMemcpyDeviceToHost);

    return image;
}

void
CudaRenderer::loadScene(SceneName scene) {
    sceneName = scene;
    loadCircleScene(sceneName, numCircles, position, color, radius);
}

void
CudaRenderer::setup() {

    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");

    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy

    hipMalloc(&cudaDevicePosition, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceColor, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceRadius, sizeof(float) * numCircles);
    hipMalloc(&cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height);

    hipMemcpy(cudaDevicePosition, position, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceColor, color, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceRadius, radius, sizeof(float) * numCircles, hipMemcpyHostToDevice);

    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    GlobalConstants params;
    params.sceneName = sceneName;
    params.numCircles = numCircles;
    params.imageWidth = image->width;
    params.imageHeight = image->height;
    params.position = cudaDevicePosition;
    params.color = cudaDeviceColor;
    params.radius = cudaDeviceRadius;
    params.imageData = cudaDeviceImageData;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void
CudaRenderer::allocOutputImage(int width, int height) {

    if (image)
        delete image;
    image = new Image(width, height);
}

// clearImage --
//
// Clear's the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void
CudaRenderer::clearImage() {

    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(
        (image->width + blockDim.x - 1) / blockDim.x,
        (image->height + blockDim.y - 1) / blockDim.y);

    kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);

    hipDeviceSynchronize();
}

void
CudaRenderer::render() {
	//numbers of threads per blocks (blockDim). If blockDim(X,Y) then there are X*Y threads per block
	dim3 blockDim(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y);
	//gridDim is the number of blocks. If gridDim(X,Y) then there are X*Y blocks
	dim3 gridDim((image->width + THREADS_PER_BLOCK_X - 1) / THREADS_PER_BLOCK_X,(image->height + THREADS_PER_BLOCK_Y - 1) / THREADS_PER_BLOCK_Y);
	kernelRenderCircles<<<gridDim, blockDim>>>();
	hipDeviceSynchronize();
}
